#include "elm.h"
#include "../nn/math/matrix_gpu.h"
#include "../util/matrix_processor.h"
#include "../nn/math/matrix_gpu.h"


void ELMGPU::set_dim(uint16_t n_hidden, uint16_t dim, uint16_t n_samples)
{
    N_Hidden = n_hidden;
    Dim = dim;
    N_Samples = n_samples;
}


void ELMGPU::init_nn_params()
{
    // generate the weights
    h_weight = (float*)malloc(N_Hidden * Dim * sizeof(float));
    hipMalloc(&d_weight, N_Hidden * Dim * sizeof(float));
    fill_rand_gpu(d_weight, N_Hidden * Dim * sizeof(float), hipMemcpyDeviceToHost);
    
    // generate the bais
    h_bias = (float*)malloc(N_Hidden * 1 * sizeof(float));
    hipMalloc(&d_bias, N_Hidden * 1 * sizeof(float));
    fill_rand_gpu(d_bias, N_Hidden, 1);  
}


void ELMGPU::config_nn(uint16_t n_hidden, uint16_t dim, uint16_t n_samples)
{
    set_dim(n_hidden, dim, n_samples);
    init_nn_params();
}


bool ELMGPU::train(fmat train_X, fmat train_Y, uint16_t activation)
{

    // host
    float* h_train_X = convert_matrix(train_X);
    float* h_train_Y = convert_matrix(train_Y);

    // device
    float *d_train_X;
    float *d_train_Y;
    hipMalloc((void**)&d_train_X, train_X.n_rows * train_X.n_cols * sizeof(float));
    hipMalloc((void**)&d_train_Y, train_Y.n_rows * train_Y.n_cols * sizeof(float));

    // host -> device
    hipMemcpy(d_train_X, h_train_X, train_X.n_rows * train_X.n_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_train_Y, h_train_Y, train_X.n_rows * train_X.n_cols * sizeof(float), hipMemcpyHostToDevice);

    // transpose weight
    float* d_weight_t = (float*)malloc(N_Hidden * Dim * sizeof(float));
    cublas_sgeam(d_weight, d_weight_t, N_Hidden, Dim);
    // train_X * weight.t()
    // float* d_hidden = (float*)malloc(train_X.n_rows * N_Hidden * sizeof(float));
    // cublas_mmul(d_train_X, d_weight_t, d_hidden, train_X.n_rows, train_X.n_cols, N_Hidden);

    hipFree(d_train_X);
    hipFree(d_train_Y);
    hipFree(d_weight_t);
    // hipFree(d_hidden);

    return true;
}


bool ELMGPU::test(float *h_test_X, float *h_test_Y, uint16_t activation)
{
    return true;
}

