#include "DeviceExampleSet.h"


    __device__ void adjust(float &x, const Range &from, const Range &to);

    DeviceExampleSet::DeviceExampleSet(uint size, uint numIn, uint numOut)
                     :ExampleSet(size, numIn, numOut)
    {
        HostExampleSet set(size, numIn, numOut);
        copyToDevice(set);
    }


    DeviceExampleSet::copyToDevice(const HostExampleSet &set)
    {
        stepBlocks = (size * step) / TPB + 1;
        outBlocks = (size * numOut) / TPB + 1;

        hipMalloc(&input, size * step * sizeof(float));
        hipMalloc(&output, size * numOut * sizeof(float));
        hipMalloc(&stat, step * sizeof(Stat));

        hipMemcpy(input, set.getInput(), size * step * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(stat, set.getStat(), step* sizeof(Stat), hipMemcpyHostToDevice);
    }
