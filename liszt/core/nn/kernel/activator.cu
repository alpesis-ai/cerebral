#include "hip/hip_runtime.h"
#include "activator.h"


void sigmoid_gpu(const float* A, float* B, int M)
{
    sigmoidKernel<<<N_BLOCKS(M), THREADS_PER_BLOCK>>>(A, B, M);
}


void sigmoidOutput_gpu(const float* A, float* B, int M, int N)
{
    sigmoidKernel<<<N_BLOCKS(M), THREADS_PER_BLOCK>>>(A, B, M*N);
}


void sigmoidGrad_gpu(const float* A, float* B, int M)
{
    sigmoidGradKernel<<<N_BLOCKS(M), THREADS_PER_BLOCK>>>(A, B, M);
}


void tanh_gpu(const float* A, float* B, int M)
{
    tanhKernel<<<N_BLOCKS(M), THREADS_PER_BLOCK>>>(A, B, M);
}


void sechSq_gpu(const float* A, float* B, int M)
{
    sechSqKernel<<<N_BLOCKS(M), THREADS_PER_BLOCK>>>(A, B, M);
}


void softmax_gpu(const float* A, float* B, int M)
{
    softmaxKernel<<<N_BLOCKS(M), THREADS_PER_BLOCK>>>(A, B, M);
}


__global__ void sigmoidKernel(const float* A, float* B, int M)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < M)
    {
        B[i] = 1 / (1 + exp(-A[i]));
    }
}


__global__ void sigmoidGradKernel(const float* A, float* B, int M)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < M)
    {
        float a = 1 / (1 + exp(-A[i]));
        B[i] = a * (1 - a);
    }
}


__global__ void tanhKernel(const float* A, float* B, int M)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < M)
    {
        B[i] = tanh(A[i]);
    }
}


__global__ void sechSqKernel(const float* A, float* B, int M)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < M)
    {
        float c = cosh(A[i]);
        B[i] = 1 / (c * c);
    }
}


__global__ void softmaxKernel(const float* A, float* B, int M, int N)
{
    // the idiot-proof un-optimised algorithm
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < M)
    {
        float total = 0;
        for (int j = 0; j < N; j++)
            total += exp(A[IDX2C(i, j, M)]);

        for (int j = 0; j < N; j++)
            B[IDX2C(i, j, M)] = exp(A[IDX2C(i, j, M)]) / total;
    }
}
