#include "hip/hip_runtime.h"
#include "initializer.h"

void fill(int *d_a, int n)
{
    int nThreadsPerBlock = 1024;
    int nBlocks = n / nThreadsPerBlock + ((n % nThreadsPerBlock) ? 1:0);

    fillKernel<<<nBlocks, nThreadsPerBlock>>>(d_a, n);
}


void randomInit(float* theta, int in, int out, int M)
{
    randomInitKernel<<<N_BLOCKS(M), THREADS_PER_BLOCK>>>(theta, in, out, M);
}


__global__ void fillKernel(int *a, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n)
    {
        a[tid] = 0;
    }
}


__global__ void randomInitKernel(float* theta, int in, int out, int M)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < M)
    {
        float epsilon = (1 / sqrt((float)in));
        theta[i] = theta[i] * 2 * epsilon - epsilon;
    }
}
