#include "hip/hip_runtime.h"
#include "cost.h"


void negLnMax_gpu(float* h, float* y, float* J, int M)
{
    negLnMaxKernel<<<N_BLOCKS(M), THREADS_PER_BLOCK>>>(h, y, J, M);
}


void crossEntropy_gpu(float* h, float* y, float* J, int M)
{
    crossEntropyKernel<<<N_BLOCKS(M), THREADS_PER_BLOCK>>>(h, y, J, M);
}


__global__ void negLnMaxKernel(float* h, float* y, float* J, int M)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < M)
    {
        J[i] = -y[i] * log(h[i]) - (1 - y[i]) * log(1 - h[i]);
    }
}


__global__ void crossEntropyKernel(float* h, float* y, float* J, int M)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < M)
    {
        J[i] = -y[i] * log(h[i]);
    }
}
